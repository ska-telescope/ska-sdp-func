#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */
  
#include<cmath>
#include "utility/sdp_device_wrapper.h"


template<
	typename inType,
	typename visType2
>
__global__ void rfi_flagger(
		const int num_time,
		const int num_baselines,
		const int num_polarisations,
		const int num_freqs,
		const int seqlen,
		const int* sequence_lengths,
		const visType2* const __restrict__ visibilities,
		const inType* const __restrict__ thresholds,
		int*  flags)
{
	
	float current_threshold = 0;
   	float tmp_sum=0.0;
    	uint64_t did=0;
	
    	did=blockIdx.x *num_freqs + threadIdx.x; 

	__shared__ inType block[256];
	__shared__ int s_flags[256];

	if(threadIdx.x<num_freqs)
	{

		for(int bid=0;bid<num_baselines;bid++)
		{

			did=blockIdx.y*num_freqs*num_polarisations*num_baselines+bid*num_freqs*num_polarisations+threadIdx.x*num_polarisations+blockIdx.x;
			block[threadIdx.x]=abs(visibilities[did].x);
			s_flags[threadIdx.x]=0;
			__syncthreads();
			current_threshold=thresholds[0] * sequence_lengths[0];

			if(block[threadIdx.x]>current_threshold)
				s_flags[threadIdx.x]=1;
			__syncthreads();
        		for (int k = 1; k < seqlen; k++)
			{
            			current_threshold = thresholds[k] * sequence_lengths[k];
				if(threadIdx.x+sequence_lengths[k]<num_freqs)
				{
					tmp_sum=block[threadIdx.x]+block[threadIdx.x+ (int)sequence_lengths[k]/2 ];
				}
				
				__syncthreads();
				
				if(threadIdx.x+sequence_lengths[k]<num_freqs)
				{
					block[threadIdx.x]=tmp_sum;
					tmp_sum=0.0;
				}
				__syncthreads();
				if(block[threadIdx.x]>current_threshold)
				{
					for(int m=threadIdx.x;m<threadIdx.x+sequence_lengths[k];m++)
					{
						s_flags[m]=1;
					}
				}
				
			}

			__syncthreads();
			flags[did]=s_flags[threadIdx.x];
			__syncthreads();
		}
	}
}

SDP_CUDA_KERNEL(rfi_flagger<float,float2>)
SDP_CUDA_KERNEL(rfi_flagger<double,double2>)
	
