#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */
  
#include "utility/sdp_device_wrapper.h"

template<typename T>
__global__ void rfi_flagger(const  int num_time,
		const  int num_freqs,
		const int seqlen,
		const int* sequence_lengths,
	       	const T* const __restrict__ spectrogram,
		const T*const __restrict__ thresholds,
		int*  flags)
{
	
	float current_threshold = 0;
   	float tmp_sum=0.0;
    	uint64_t did=0;
	
    	did=blockIdx.x *num_freqs + threadIdx.x; 

	__shared__ float block[256];
	__shared__ int s_flags[256];

	if(threadIdx.x<num_freqs)
	{
		block[threadIdx.x]=spectrogram[did];
		s_flags[threadIdx.x]=0;
		__syncthreads();
		current_threshold=thresholds[0] * sequence_lengths[0];

		if(block[threadIdx.x]>current_threshold)
			s_flags[threadIdx.x]=1;
		__syncthreads();
        	for (int k = 1; k < seqlen; k++)
		{
            			current_threshold = thresholds[k] * sequence_lengths[k];
				if(threadIdx.x+sequence_lengths[k]<num_freqs)
				{
					tmp_sum=block[threadIdx.x]+block[threadIdx.x+ (int)sequence_lengths[k]/2 ];
				}
				
				__syncthreads();
				
				if(threadIdx.x+sequence_lengths[k]<num_freqs)
				{
					block[threadIdx.x]=tmp_sum;
					tmp_sum=0.0;
				}
				__syncthreads();
				if(block[threadIdx.x]>current_threshold)
				{
					for(int m=threadIdx.x;m<threadIdx.x+sequence_lengths[k];m++)
					{
						s_flags[m]=1;
					}
				}
				
		}


		__syncthreads();
		flags[did]=s_flags[threadIdx.x];
		__syncthreads();
		
	}
}

SDP_CUDA_KERNEL(rfi_flagger<float>)
	
