#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */

#include "ska-sdp-func/utility/sdp_device_wrapper.h"
#include "ska-sdp-func/numeric_functions/sdp_fft_convolution.h"
#include "ska-sdp-func/utility/sdp_mem.h"

#include <hip/hip_complex.h>
#include <hip/hip_bf16.h>

// #define INDEX_2D(N2, N1, I2, I1)    (N1 * I2 + I1)

// create a copy of a complex number using only its real part
template<typename CT, typename T>
__global__ void create_copy_real(
    const CT* in,
    int64_t size,
    T* out){

    }

template<>
__global__ void create_copy_real<hipDoubleComplex, double>(
    const hipDoubleComplex* in,
    int64_t size,
    double* out
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        out[i] = hipCreal(in[i]);
    }
}

template<>
__global__ void create_copy_real<hipFloatComplex, float>(
    const hipFloatComplex* in,
    int64_t size,
    float* out
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        out[i] = hipCrealf(in[i]);
    }
}

SDP_CUDA_KERNEL(create_copy_real<hipDoubleComplex, double>);
SDP_CUDA_KERNEL(create_copy_real<hipFloatComplex, float>);


// create the CLEAN beam
template<typename T, typename CT>
__global__ void create_cbeam(
    const T* cbeam_details,
    int16_t psf_dim,
    CT* cbeam
) {

}

template<>
__global__ void create_cbeam<double, hipDoubleComplex>(
    const double* cbeam_details,
    int16_t psf_dim,
    hipDoubleComplex* cbeam
) {
    // Fit a Gaussian to the main lobe of the PSF based on the parameters passed

    double A = 1;
    double x0 = (psf_dim / 2);
    double y0 = (psf_dim / 2);
    double sigma_X = cbeam_details[0];
    double sigma_Y = cbeam_details[1];
    double theta = (M_PI / 180) * cbeam_details[2];

    double a = pow(cos(theta), 2) / (2 * pow(sigma_X, 2)) + pow(sin(theta), 2) / (2 * pow(sigma_Y, 2));
    double b = sin(2 * theta) / (4 * pow(sigma_X, 2)) - sin(2 * theta) / (4 * pow(sigma_Y, 2));
    double c = pow(sin(theta), 2) / (2 * pow(sigma_X, 2)) + pow(cos(theta), 2) / (2 * pow(sigma_Y, 2));

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int size = psf_dim * psf_dim;

    if (i < size){ 
        int x = i / psf_dim;
        int y = i % psf_dim;

        double component = A * exp(-(a * pow(x - x0, 2) + 2 * b * (x - x0) * (y - y0) + c * pow(y - y0, 2)));
        cbeam[i] = make_hipDoubleComplex(component, 0);
    }
}

template<>
__global__ void create_cbeam<float, hipFloatComplex>(
    const float* cbeam_details,
    int16_t psf_dim,
    hipFloatComplex* cbeam
) {
    // Fit a Gaussian to the main lobe of the PSF based on the parameters passed

    float A = 1;
    float x0 = (psf_dim / 2);
    float y0 = (psf_dim / 2);
    float sigma_X = cbeam_details[0];
    float sigma_Y = cbeam_details[1];
    float theta = (M_PI / 180) * cbeam_details[2];

    float a = pow(cos(theta), 2) / (2 * pow(sigma_X, 2)) + pow(sin(theta), 2) / (2 * pow(sigma_Y, 2));
    float b = sin(2 * theta) / (4 * pow(sigma_X, 2)) - sin(2 * theta) / (4 * pow(sigma_Y, 2));
    float c = pow(sin(theta), 2) / (2 * pow(sigma_X, 2)) + pow(cos(theta), 2) / (2 * pow(sigma_Y, 2));

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int size = psf_dim * psf_dim;

    if (i < size){ 
        int x = i / psf_dim;
        int y = i % psf_dim;

        float component = A * exp(-(a * pow(x - x0, 2) + 2 * b * (x - x0) * (y - y0) + c * pow(y - y0, 2)));
        cbeam[i] = make_hipFloatComplex(component, 0);
    }
}

SDP_CUDA_KERNEL(create_cbeam<double, hipDoubleComplex>);
SDP_CUDA_KERNEL(create_cbeam<float, hipFloatComplex>);


// find the maximum value in a list using reduction
template<typename T>
__global__ void find_maximum_value(
            const T *input,
            int *index_in,
            T *output,
            int *index_out,
            bool init_idx,
            bool thresh_reached)
{
    // check if flux threshold has been reached
    if (thresh_reached == false){
        __shared__ T max_values[256];
        __shared__ int max_indices[256];

        int64_t tid = threadIdx.x;
        int64_t i = blockIdx.x * (blockDim.x) + threadIdx.x;

        // Load input elements into shared memory
        max_values[tid] = input[i];
        // if index array has already been initialised then load it
        if(init_idx == true){
            max_indices[tid] = index_in[i];

        }
        // if it hasn't, initialise it.
        else{

            max_indices[tid] = i;

        }
        __syncthreads();


        // Perform reduction in shared memory
        for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (tid < stride) {
                if (max_values[tid] < max_values[tid + stride]) {
                    max_values[tid] = max_values[tid + stride];
                    max_indices[tid] = max_indices[tid + stride];
                }
            }
            __syncthreads();
        }

        // Write the final result to output
        if (tid == 0) {
            output[blockIdx.x] = max_values[0];
            index_out[blockIdx.x] = max_indices[0];
        }
    }

}

SDP_CUDA_KERNEL(find_maximum_value<double>);
SDP_CUDA_KERNEL(find_maximum_value<float>);
SDP_CUDA_KERNEL(find_maximum_value<__hip_bfloat16>);


// add a component to the CLEAN component list
template<typename T>
__global__ void add_clean_comp(
            T* clean_comp,
            int* max_idx_flat,
            T* loop_gain,
            T* highest_value,
            T* threshold,
            bool thresh_reached
){
    // check threshold
    if (highest_value[0] > threshold[0] && thresh_reached == false){
        
        // Add fraction of maximum to clean components list
        clean_comp[max_idx_flat[0]] = clean_comp[max_idx_flat[0]] + (loop_gain[0] * highest_value[0]);

    }
    // if threshold reached, set flag
    else{
        thresh_reached = true;
    }

}

SDP_CUDA_KERNEL(add_clean_comp<double>);
SDP_CUDA_KERNEL(add_clean_comp<float>);
SDP_CUDA_KERNEL(add_clean_comp<__hip_bfloat16>);


// subtract the psf from the residual image
template<typename T>
__global__ void subtract_psf(
            int64_t dirty_img_dim, 
            int64_t psf_dim, 
            T* loop_gain, 
            int* max_idx_flat, 
            T* highest_value, 
            const T* psf, 
            T* residual,
            T* clean_comp,
            T* skymodel,
            T* threshold
) {
            
}

template<>
__global__ void subtract_psf(
            int64_t dirty_img_dim, 
            int64_t psf_dim, 
            double* loop_gain, 
            int* max_idx_flat, 
            double* highest_value, 
            const double* psf, 
            double* residual,
            double* clean_comp,
            double* skymodel,
            double* threshold) {

    // check threshold
    if (highest_value[0] > threshold[0]){

        int64_t dirty_img_size = dirty_img_dim * dirty_img_dim;
        // int64_t psf_size = psf_dim * psf_dim;

        // get x and y from flat index
        int max_idx_x = max_idx_flat[0] / dirty_img_dim;
        int max_idx_y = max_idx_flat[0] % dirty_img_dim;

        // Identify start position of PSF window to subtract from residual
        int64_t psf_x_start = dirty_img_dim - max_idx_x;
        int64_t psf_y_start = dirty_img_dim - max_idx_y;

        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

        // check thread is in bounds
        if (i < dirty_img_size){

            // Compute the x and y coordinates in the dirty image
            int64_t x_dirty = i / dirty_img_dim;
            int64_t y_dirty = i % dirty_img_dim;

            // Compute the x and y coordinates in the psf
            int64_t x_psf = x_dirty + psf_x_start;
            int64_t y_psf = y_dirty + psf_y_start;

            // // get flat index for dirty image
            // int64_t dirty_img_flat_idx = x_dirty * dirty_img_dim + y_dirty;

            // get flat index for psf
            int64_t psf_flat_idx = x_psf * psf_dim + y_psf;

            // Subtract the PSF contribution from the residual
            double inter = __dmul_rn(loop_gain[0], highest_value[0]);
            inter = __dmul_rn(inter, psf[psf_flat_idx]);
            residual[i] =  __dsub_rn(residual[i],inter);
           
            // residual[i] = residual[i] - (loop_gain[0] * highest_value[0] * psf[psf_flat_idx]);
        }
    }
    else{
        return;
    }
}

template<>
__global__ void subtract_psf(
            int64_t dirty_img_dim, 
            int64_t psf_dim, 
            float* loop_gain, 
            int* max_idx_flat, 
            float* highest_value, 
            const float* psf, 
            float* residual,
            float* clean_comp,
            float* skymodel,
            float* threshold) {

    // check threshold
    if (highest_value[0] > threshold[0]){

        int64_t dirty_img_size = dirty_img_dim * dirty_img_dim;
        // int64_t psf_size = psf_dim * psf_dim;

        // get x and y from flat index
        int max_idx_x = max_idx_flat[0] / dirty_img_dim;
        int max_idx_y = max_idx_flat[0] % dirty_img_dim;

        // Identify start position of PSF window to subtract from residual
        int64_t psf_x_start = dirty_img_dim - max_idx_x;
        int64_t psf_y_start = dirty_img_dim - max_idx_y;

        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

        // check thread is in bounds
        if (i < dirty_img_size){

            // Compute the x and y coordinates in the dirty image
            int64_t x_dirty = i / dirty_img_dim;
            int64_t y_dirty = i % dirty_img_dim;

            // Compute the x and y coordinates in the psf
            int64_t x_psf = x_dirty + psf_x_start;
            int64_t y_psf = y_dirty + psf_y_start;

            // // get flat index for dirty image
            // int64_t dirty_img_flat_idx = x_dirty * dirty_img_dim + y_dirty;

            // get flat index for psf
            int64_t psf_flat_idx = x_psf * psf_dim + y_psf;

            // Subtract the PSF contribution from the residual
            float inter = __fmul_rn(loop_gain[0], highest_value[0]);
            inter = __fmul_rn(inter, psf[psf_flat_idx]);
            residual[i] =  __fsub_rn(residual[i],inter);
           
            // residual[i] = residual[i] - (loop_gain[0] * highest_value[0] * psf[psf_flat_idx]);
        }
    }
}

template<>
__global__ void subtract_psf(
            int64_t dirty_img_dim, 
            int64_t psf_dim, 
            __hip_bfloat16* loop_gain, 
            int* max_idx_flat, 
            __hip_bfloat16* highest_value, 
            const __hip_bfloat16* psf, 
            __hip_bfloat16* residual,
            __hip_bfloat16* clean_comp,
            __hip_bfloat16* skymodel,
            __hip_bfloat16* threshold) {

    // check threshold
    if (highest_value[0] > threshold[0]){

        int64_t dirty_img_size = dirty_img_dim * dirty_img_dim;
        // int64_t psf_size = psf_dim * psf_dim;

        // get x and y from flat index
        int max_idx_x = max_idx_flat[0] / dirty_img_dim;
        int max_idx_y = max_idx_flat[0] % dirty_img_dim;

        // Identify start position of PSF window to subtract from residual
        int64_t psf_x_start = dirty_img_dim - max_idx_x;
        int64_t psf_y_start = dirty_img_dim - max_idx_y;

        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

        // check thread is in bounds
        if (i < dirty_img_size){

            // Compute the x and y coordinates in the dirty image
            int64_t x_dirty = i / dirty_img_dim;
            int64_t y_dirty = i % dirty_img_dim;

            // Compute the x and y coordinates in the psf
            int64_t x_psf = x_dirty + psf_x_start;
            int64_t y_psf = y_dirty + psf_y_start;

            // // get flat index for dirty image
            // int64_t dirty_img_flat_idx = x_dirty * dirty_img_dim + y_dirty;

            // get flat index for psf
            int64_t psf_flat_idx = x_psf * psf_dim + y_psf;

            // Subtract the PSF contribution from the residual
            // __hip_bfloat16 inter = __hmul_rn(loop_gain[0], highest_value[0]);
            // inter = __hmul_rn(inter, psf[psf_flat_idx]);
            // residual[i] =  __hsub_rn(residual[i],inter);
           
            residual[i] = residual[i] - (loop_gain[0] * highest_value[0] * psf[psf_flat_idx]);
        }
    }
    else{
        return;
    }
}

SDP_CUDA_KERNEL(subtract_psf<double>);
SDP_CUDA_KERNEL(subtract_psf<float>);
SDP_CUDA_KERNEL(subtract_psf<__hip_bfloat16>);


// add the final residual image to the skymodel
template<typename T>
__global__ void add_residual(
            T* in,
            int64_t size,
            T* out
){
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < (size)){
        out[i] = out[i] + in[i];
    }
}

SDP_CUDA_KERNEL(add_residual<double>);
SDP_CUDA_KERNEL(add_residual<float>);


// create a copy of a real value as a complex value with imaginary part set to 0
template<typename T, typename CT>
__global__ void create_copy_complex(
    const T* in,
    int64_t size,
    CT* out
) {

}

template<>
__global__ void create_copy_complex<double, hipDoubleComplex>(
    const double* in,
    int64_t size,
    hipDoubleComplex* out
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        out[i] = make_hipDoubleComplex(in[i], 0);
    }
}

template<>
__global__ void create_copy_complex<float, hipFloatComplex>(
    const float* in,
    int64_t size,
    hipFloatComplex* out
) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        out[i] = make_hipFloatComplex(in[i], 0);
    }
}

SDP_CUDA_KERNEL(create_copy_complex<double, hipDoubleComplex>);
SDP_CUDA_KERNEL(create_copy_complex<float, hipFloatComplex>);


// convert a number from bfloat16 to float or double precision
template<typename T>
__global__ void convert_from_bfloat(
    const __hip_bfloat16* in,
    int64_t size,
    T* out
){

}

template<>
__global__ void convert_from_bfloat<float>(
    const __hip_bfloat16* in,
    int64_t size,
    float* out
){
    
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        out[i] = __bfloat162float(in[i]);
         
    }
}

template<>
__global__ void convert_from_bfloat<double>(
    const __hip_bfloat16* in,
    int64_t size,
    double* out
){
        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        out[i] = in[i];
         
    }
}

SDP_CUDA_KERNEL(convert_from_bfloat<double>);
SDP_CUDA_KERNEL(convert_from_bfloat<float>);


// cpnvert a double or single precision number to bfloat16
template<typename T>
__global__ void convert_to_bfloat(
    const T* in,
    int64_t size,
    __hip_bfloat16* out
){

}

template<>
__global__ void convert_to_bfloat<double>(
    const double* in,
    int64_t size,
    __hip_bfloat16* out
){
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        out[i] = __double2bfloat16(in[i]);
         
    }

}

template<>
__global__ void convert_to_bfloat<float>(
    const float* in,
    int64_t size,
    __hip_bfloat16* out
){
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        out[i] = __float2bfloat16(in[i]);
        
    }

}

SDP_CUDA_KERNEL(convert_to_bfloat<double>);
SDP_CUDA_KERNEL(convert_to_bfloat<float>);


// copy one value to gpu, can convert to bfloat from float or double precision
// needed to convert loop gain and threshold to bfloat16 for correct use in maths with bfloat16 data 
template<typename T, typename OT>
__global__ void copy_var_gpu(
    T in,
    OT* out
){

}

template<>
__global__ void copy_var_gpu<double, double>(
    double in,
    double* out
){
    out[0] = in;
}

template<>
__global__ void copy_var_gpu<float, float>(
    float in,
    float* out
){
    out[0] = in;
}

template<>
__global__ void copy_var_gpu<double, __hip_bfloat16>(
    double in,
    __hip_bfloat16* out
){
    out[0] = __double2bfloat16(in);
}

template<>
__global__ void copy_var_gpu<float, __hip_bfloat16>(
    float in,
    __hip_bfloat16* out
){
    out[0] = __float2bfloat16(in);
}

SDP_CUDA_KERNEL(copy_var_gpu<double, double>);
SDP_CUDA_KERNEL(copy_var_gpu<float, float>);
SDP_CUDA_KERNEL(copy_var_gpu<double, __hip_bfloat16>);
SDP_CUDA_KERNEL(copy_var_gpu<float, __hip_bfloat16>);

// // max finding atomic experiment
// __device__ __forceinline__ void my_atomic_max(double* addr, double value)
// {

//     unsigned long long int* laddr = (unsigned long long int*)(addr);
//     unsigned long long int assumed, old_ = *laddr;
//     do
//     {
//         assumed = old_;
//         old_ = atomicCAS(laddr,
//                 assumed,
//                 __double_as_longlong(max(value,
//                 __longlong_as_double(assumed)))
//                 );
//     }
//     while (assumed != old_);

// }


// typedef union  {
//   double floats[2];                 // floats[0] = lowest
//   int ints[2];                     // ints[1] = lowIdx
//   unsigned long long int ulong;    // for atomic update
// } my_atomics;

// __device__ my_atomics test;

// __device__ unsigned long long int my_atomicMin(unsigned long long int* address, double val1, int val2)
// {
//     my_atomics loc, loctest;
//     loc.floats[0] = val1;
//     loc.ints[1] = val2;
//     loctest.ulong = *address;
//     while (loctest.floats[0] <  val1) 
//       loctest.ulong = atomicCAS(address, loctest.ulong,  loc.ulong);
//     return loctest.ulong;
// }



// __global__ void find_maximum_value_atomic(
//             double* residual, 
//             int64_t dirty_img_size,
//             double* highest_value, 
//             int* max_idx_flat){

//     __shared__ double shared_values[256];
//     __shared__ int shared_indices[256];

//     int tid = threadIdx.x;
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

//     // Load input elements into shared memory
//     shared_values[tid] = residual[i];
//     shared_indices[tid] = i;

//     __syncthreads();

//     // Perform reduction
//     for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
//         if (tid < stride) {
//             if (shared_values[tid + stride] > shared_values[tid]) {
//                 shared_values[tid] = shared_values[tid + stride];
//                 shared_indices[tid] = shared_indices[tid + stride];
//             }
//         }
//         __syncthreads();
//     }

//     // Update global maximum value and index
//     if (tid == 0) {

//         // my_atomic_max(&highest_value[0],shared_values[0]);

//         my_atomicMin(&(test.ulong), highest_value[0],max_idx_flat[0]);
//     }

//     highest_value[0] = *test.floats;
//     // highest_value[1] = *test.ints;


//     // // Write the final result to output
//     // if (tid == 0) {
//     //     output[blockIdx.x] = max_values[0];
//     //     index[blockIdx.x] = max_indices[0];
//     // }


// }
// SDP_CUDA_KERNEL(find_maximum_value_atomic)








// previous code
// __device__ __forceinline__ void my_atomic_max(double* addr, double value)
// {

//     unsigned long long int* laddr = (unsigned long long int*)(addr);
//     unsigned long long int assumed, old_ = *laddr;
//     do
//     {
//         assumed = old_;
//         old_ = atomicCAS(laddr,
//                 assumed,
//                 __double_as_longlong(max(value,
//                 __longlong_as_double(assumed)))
//                 );
//     }
//     while (assumed != old_);

// }


// __device__ void find_maximum_value(
//             double* residual, 
//             int64_t dirty_img_size,
//             double* highest_value, 
//             int* max_idx_flat,
//             double* skymodel){

//     __shared__ double shared_values[256];
//     __shared__ int shared_indices[256];

//     int tid = threadIdx.x;
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

//     // Load input elements into shared memory
//     shared_values[tid] = residual[i];
//     shared_indices[tid] = i;

//     __syncthreads();

//     // Perform reduction
//     for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
//         if (tid < stride) {
//             if (shared_values[tid + stride] > shared_values[tid]) {
//                 shared_values[tid] = shared_values[tid + stride];
//                 shared_indices[tid] = shared_indices[tid + stride];
//             }
//         }
//         __syncthreads();
//     }

//     // Update global maximum value and index
//     if (tid == 0) {
//         // if (shared_values[0] > *highest_value){
//         //     *highest_value = shared_values[0];
//         //     *max_idx_flat = shared_indices[0];

//         // }

//         skymodel[blockIdx.x] = shared_values[0];
//         // my_atomic_max(highest_value,shared_values[0]);
//         // *max_idx_flat = shared_indices[0];

//     }


// }

// __device__ void subtract_psf(
//             int64_t dirty_img_dim, 
//             int64_t psf_dim, 
//             int64_t psf_x_start, 
//             int64_t psf_y_start, 
//             double loop_gain, 
//             double highest_value, 
//             const double* psf, 
//             double* residual,
//             double* skymodel) {

//     int64_t dirty_img_size = dirty_img_dim * dirty_img_dim;
//     int64_t psf_size = psf_dim * psf_dim;

//     int i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < dirty_img_size){

//         // Compute the x and y coordinates in the dirty image
//         int64_t x_dirty = i / dirty_img_dim;
//         int64_t y_dirty = i % dirty_img_dim;

//         // Compute the x and y coordinates in the psf
//         int64_t x_psf = x_dirty + psf_x_start;
//         int64_t y_psf = y_dirty + psf_y_start;

//         // // get flat index for dirty image
//         // int64_t dirty_img_flat_idx = x_dirty * dirty_img_dim + y_dirty;

//         // get flat index for psf
//         int64_t psf_flat_idx = x_psf * psf_dim + y_psf;

//         // Subtract the PSF contribution from the residual
//         residual[i] -= (loop_gain * highest_value * psf[psf_flat_idx]);

//         // skymodel[i] = psf[psf_flat_idx];

//     }
// }


// __device__ void create_copy_complex(
//     const double* in,
//     int64_t size,
//     hipDoubleComplex* out
// ) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (i < size){
//         out[i] = make_hipDoubleComplex(in[i], 0);
//     }
// }

// __device__ void copy_to_skymodel(
//     const double* in,
//     int64_t size,
//     double* out
// ) {
//     for (int i = 0; i < size; i++){
//         out[i] = in[i];
//     }
// }


// __global__ void Hogbom_clean(
//             const double* psf,
//             const double loop_gain,
//             const double threshold,
//             const double cycle_limit,
//             double* residual,
//             const int64_t dirty_img_dim,
//             const int64_t psf_dim,
//             double* clean_comp,
//             hipDoubleComplex* clean_comp_complex,
//             double* skymodel
// ){
//     // calculate useful shapes and sizes
//     int64_t dirty_img_size = dirty_img_dim * dirty_img_dim;
//     int64_t psf_size = psf_dim * psf_dim;

//     // set up some loop variables
//     int cur_cycle = 0;
//     bool stop = 0;

//     // CLEAN loop executes while the stop conditions (threshold and cycle limit) are not met
//     while (cur_cycle < 1 && !stop) {
//         // Find index and value of the maximum value in residual
//         double highest_value = 0.0;
//         int max_idx_flat = 0;

//         find_maximum_value(residual, dirty_img_size, &highest_value, &max_idx_flat, clean_comp);
//         copy_to_skymodel(clean_comp, dirty_img_size, skymodel);
//         // find_maximum_value(clean_comp, dirty_img_size, &highest_value, &max_idx_flat, skymodel);


//         // Check maximum value against threshold
//         if (highest_value < threshold) {
//             stop = true;
//             break;
//         }

//         // int max_idx_flat_test = 256 *1024 + 256;
//         // skymodel[0] = (double)max_idx_flat;
//         // skymodel[1] = (double)max_idx_flat_test;

//         // get x and y from flat index
//         int max_idx_x = max_idx_flat / dirty_img_dim;
//         int max_idx_y = max_idx_flat % dirty_img_dim;

//         // highest_value = 10.0;

//         // skymodel[2] = (double)max_idx_x;
//         // skymodel[3] = (double)max_idx_y;
//         // skymodel[4] = highest_value;

//         // Add fraction of maximum to clean components list
//         clean_comp[max_idx_flat] += (loop_gain * highest_value);

//         // Identify start position of PSF window to subtract from residual
//         int64_t psf_x_start = dirty_img_dim - max_idx_x;
//         int64_t psf_y_start = dirty_img_dim - max_idx_y;

//         subtract_psf(dirty_img_dim, psf_dim, psf_x_start, psf_y_start, loop_gain, highest_value, psf, residual,skymodel);

//         cur_cycle += 1;
//     }

//     // copy_to_skymodel(clean_comp, dirty_img_size, skymodel);

//     // Convolution code only works with complex input, so make clean components and clean beam complex
//     create_copy_complex(clean_comp, dirty_img_size, clean_comp_complex);

// }

// SDP_CUDA_KERNEL(Hogbom_clean);


