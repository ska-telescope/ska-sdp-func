#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */

#include <thrust/complex.h>

#include "ska-sdp-func/fourier_transforms/private_pswf.h"
#include "ska-sdp-func/grid_data/sdp_gridder_utils.h"
#include "ska-sdp-func/math/sdp_math_macros.h"
#include "ska-sdp-func/utility/sdp_cuda_atomics.h"
#include "ska-sdp-func/utility/sdp_device_wrapper.h"
#include "ska-sdp-func/utility/sdp_mem_view.h"

using thrust::complex;


template<typename OUT_TYPE, typename IN1_TYPE, typename IN2_TYPE>
__global__ void sdp_gridder_accum_scale_array(
        sdp_MemViewGpu<OUT_TYPE, 2> out,
        const sdp_MemViewGpu<const IN1_TYPE, 2> in1,
        const sdp_MemViewGpu<const IN2_TYPE, 2> in2,
        const int exponent,
        const int use_in2
)
{
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= out.shape[0] || j >= out.shape[1]) return;
    if (use_in2)
    {
        if (exponent == 1)
        {
            out(i, j) += (IN2_TYPE) in1(i, j) * in2(i, j);
        }
        else
        {
            out(i, j) += (IN2_TYPE) in1(i, j) * pow(in2(i, j), exponent);
        }
    }
    else
    {
        out(i, j) += in1(i, j);
    }
}


template<typename T>
__global__ void sdp_gridder_grid_correct_pswf(
        int image_size,
        double theta,
        double w_step,
        double shear_u,
        double shear_v,
        const sdp_MemViewGpu<const double, 1> pswf,
        const double* const __restrict__ pswf_n_coeff,
        double pswf_n_c,
        sdp_MemViewGpu<T, 2> facet,
        int facet_offset_l,
        int facet_offset_m
)
{
    const int64_t il = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t im = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t num_l = facet.shape[0];
    const int64_t num_m = facet.shape[1];
    if (il >= num_l || im >= num_m) return;
    const int pl = il - num_l / 2 + facet_offset_l;
    const int pm = im - num_m / 2 + facet_offset_m;
    const double l_ = pl * theta / image_size;
    const double m_ = pm * theta / image_size;
    const double n_ = lm_to_n(l_, m_, shear_u, shear_v);
    const double pswf_l = pswf(pl + image_size / 2);
    const double pswf_m = pswf(pm + image_size / 2);
    const double pswf_n_x = fabs(n_ * 2.0 * w_step);
    const double pswf_n = (pswf_n_x < 1.0) ?
                sdp_pswf_aswfa(0, 0, pswf_n_c, pswf_n_coeff, pswf_n_x) : 1.0;
    const double scale = 1.0 / (pswf_l * pswf_m * pswf_n);
    facet(il, im) *= (T) scale;
}


template<typename T>
__global__ void sdp_gridder_grid_correct_w_stack(
        int image_size,
        double theta,
        double w_step,
        double shear_u,
        double shear_v,
        sdp_MemViewGpu<T, 2> facet,
        int facet_offset_l,
        int facet_offset_m,
        int w_offset,
        int inverse
)
{
    const int64_t il = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t im = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t num_l = facet.shape[0];
    const int64_t num_m = facet.shape[1];
    if (il >= num_l || im >= num_m) return;
    const int pl = il - num_l / 2 + facet_offset_l;
    const int pm = im - num_m / 2 + facet_offset_m;
    const double l_ = pl * theta / image_size;
    const double m_ = pm * theta / image_size;
    const double n_ = lm_to_n(l_, m_, shear_u, shear_v);
    const double phase = 2.0 * M_PI * w_step * n_;
    complex<double> w = complex<double>(cos(phase), sin(phase));
    w = pow(w, w_offset);
    w = !inverse ? 1.0 / w : w;
    facet(il, im) *= (T) w;
}


template<typename OUT_TYPE, typename IN1_TYPE, typename IN2_TYPE>
__global__ void sdp_gridder_scale_inv_array(
        sdp_MemViewGpu<OUT_TYPE, 2> out,
        const sdp_MemViewGpu<const IN1_TYPE, 2> in1,
        const sdp_MemViewGpu<const IN2_TYPE, 2> in2,
        const int exponent
)
{
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= out.shape[0] || j >= out.shape[1]) return;
    if (exponent == 1)
    {
        out(i, j) = (IN2_TYPE) in1(i, j) / in2(i, j);
    }
    else
    {
        out(i, j) = (IN2_TYPE) in1(i, j) / pow(in2(i, j), exponent);
    }
}


template<typename GRID_TYPE, typename SUBGRID_TYPE, typename FACTOR_TYPE>
__global__ void sdp_gridder_subgrid_add(
        sdp_MemViewGpu<GRID_TYPE, 2> grid,
        int offset_u,
        int offset_v,
        sdp_MemViewGpu<const SUBGRID_TYPE, 2> subgrid,
        FACTOR_TYPE factor
)
{
    const int64_t sub_size_u = subgrid.shape[0], sub_size_v = subgrid.shape[1];
    const int64_t grid_size_u = grid.shape[0], grid_size_v = grid.shape[1];
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= sub_size_u || j >= sub_size_v) return;
    int64_t i1 = i + grid_size_u / 2 - sub_size_u / 2 - offset_u;
    int64_t j1 = j + grid_size_v / 2 - sub_size_v / 2 - offset_v;
    while (i1 < 0)
    {
        i1 += grid_size_u;
    }
    while (i1 >= grid_size_u)
    {
        i1 -= grid_size_u;
    }
    while (j1 < 0)
    {
        j1 += grid_size_v;
    }
    while (j1 >= grid_size_v)
    {
        j1 -= grid_size_v;
    }
    grid(i1, j1) += subgrid(i, j) * factor;
}


template<typename GRID_TYPE, typename SUBGRID_TYPE>
__global__ void sdp_gridder_subgrid_cut_out(
        sdp_MemViewGpu<const GRID_TYPE, 2> grid,
        int offset_u,
        int offset_v,
        sdp_MemViewGpu<SUBGRID_TYPE, 2> subgrid
)
{
    const int64_t sub_size_u = subgrid.shape[0], sub_size_v = subgrid.shape[1];
    const int64_t grid_size_u = grid.shape[0], grid_size_v = grid.shape[1];
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= sub_size_u || j >= sub_size_v) return;
    int64_t i1 = i + grid_size_u / 2 - sub_size_u / 2 - offset_u;
    int64_t j1 = j + grid_size_v / 2 - sub_size_v / 2 - offset_v;
    while (i1 < 0)
    {
        i1 += grid_size_u;
    }
    while (i1 >= grid_size_u)
    {
        i1 -= grid_size_u;
    }
    while (j1 < 0)
    {
        j1 += grid_size_v;
    }
    while (j1 >= grid_size_v)
    {
        j1 -= grid_size_v;
    }
    subgrid(i, j) = grid(i1, j1);
}


template<typename T, int BLOCK_SIZE>
__device__ void warp_reduce(volatile T* smem, int thread_id)
{
    if (BLOCK_SIZE >= 64) smem[thread_id] += smem[thread_id + 32];
    if (BLOCK_SIZE >= 32) smem[thread_id] += smem[thread_id + 16];
    if (BLOCK_SIZE >= 16) smem[thread_id] += smem[thread_id + 8];
    if (BLOCK_SIZE >= 8) smem[thread_id] += smem[thread_id + 4];
    if (BLOCK_SIZE >= 4) smem[thread_id] += smem[thread_id + 2];
    if (BLOCK_SIZE >= 2) smem[thread_id] += smem[thread_id + 1];
}


template<typename T, int BLOCK_SIZE>
__global__ void sdp_gridder_sum_diff(
        sdp_MemViewGpu<const T, 1> a,
        sdp_MemViewGpu<const T, 1> b,
        T* result
)
{
    extern __shared__ T smem[];
    const int64_t n = MIN(a.shape[0], b.shape[0]);
    const int thread_id = threadIdx.x;
    const int64_t grid_size = BLOCK_SIZE * 2 * gridDim.x;
    int64_t i = blockIdx.x * (BLOCK_SIZE * 2) + thread_id;
    smem[thread_id] = (T) 0;
    while (i < n)
    {
        smem[thread_id] += a(i) - b(i);
        if (i + BLOCK_SIZE < n)
        {
            smem[thread_id] += a(i + BLOCK_SIZE) - b(i + BLOCK_SIZE);
        }
        i += grid_size;
    }
    __syncthreads();
    if (BLOCK_SIZE >= 512)
    {
        if (thread_id < 256) smem[thread_id] += smem[thread_id + 256];
        __syncthreads();
    }
    if (BLOCK_SIZE >= 256)
    {
        if (thread_id < 128) smem[thread_id] += smem[thread_id + 128];
        __syncthreads();
    }
    if (BLOCK_SIZE >= 128)
    {
        if (thread_id < 64) smem[thread_id] += smem[thread_id + 64];
        __syncthreads();
    }
    if (thread_id < warpSize) warp_reduce<T, BLOCK_SIZE>(smem, thread_id);
    if (thread_id == 0) atomicAdd(result, smem[0]);
}


template<typename UVW_TYPE>
__global__ void sdp_gridder_uvw_bounds_all(
        const sdp_MemViewGpu<const UVW_TYPE, 2> uvws,
        const double freq0_hz,
        const double dfreq_hz,
        const sdp_MemViewGpu<const int, 1> start_chs,
        const sdp_MemViewGpu<const int, 1> end_chs,
        double* uvw_min,
        double* uvw_max
)
{
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t num_uvw = uvws.shape[0];
    if (i >= num_uvw)
        return;
    const int start_ch = start_chs(i), end_ch = end_chs(i);
    if (start_ch >= end_ch)
        return;
    const double uvw[] = {uvws(i, 0), uvws(i, 1), uvws(i, 2)};
    #pragma unroll
    for (int j = 0; j < 3; ++j)
    {
        const double u0 = freq0_hz * uvw[j] / C_0;
        const double du = dfreq_hz * uvw[j] / C_0;
        if (uvw[j] >= 0)
        {
            (void)sdp_atomic_min(&uvw_min[j], u0 + start_ch * du);
            (void)sdp_atomic_max(&uvw_max[j], u0 + (end_ch - 1) * du);
        }
        else
        {
            (void)sdp_atomic_max(&uvw_max[j], u0 + start_ch * du);
            (void)sdp_atomic_min(&uvw_min[j], u0 + (end_ch - 1) * du);
        }
    }
}


// *INDENT-OFF*
SDP_CUDA_KERNEL(sdp_gridder_accum_scale_array<complex<double>, complex<double>, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_accum_scale_array<complex<float>, complex<float>, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_accum_scale_array<complex<double>, complex<float>, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_accum_scale_array<complex<float>, complex<double>, complex<double> >)

SDP_CUDA_KERNEL(sdp_gridder_grid_correct_pswf<double>)
SDP_CUDA_KERNEL(sdp_gridder_grid_correct_pswf<float>)
SDP_CUDA_KERNEL(sdp_gridder_grid_correct_pswf<complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_grid_correct_pswf<complex<float> >)

SDP_CUDA_KERNEL(sdp_gridder_grid_correct_w_stack<complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_grid_correct_w_stack<complex<float> >)

SDP_CUDA_KERNEL(sdp_gridder_scale_inv_array<complex<double>, double, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_scale_inv_array<complex<float>, float, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_scale_inv_array<complex<double>, complex<double>, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_scale_inv_array<complex<float>, complex<float>, complex<double> >)

SDP_CUDA_KERNEL(sdp_gridder_subgrid_add<complex<double>, complex<double>, double>)
SDP_CUDA_KERNEL(sdp_gridder_subgrid_add<complex<float>, complex<float>, double>)

SDP_CUDA_KERNEL(sdp_gridder_subgrid_cut_out<complex<double>, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_subgrid_cut_out<complex<float>, complex<float> >)

SDP_CUDA_KERNEL(sdp_gridder_sum_diff<int, 512>)

SDP_CUDA_KERNEL(sdp_gridder_uvw_bounds_all<double>)
SDP_CUDA_KERNEL(sdp_gridder_uvw_bounds_all<float>)
// *INDENT-ON*
