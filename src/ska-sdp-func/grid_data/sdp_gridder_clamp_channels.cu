#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */

#include "ska-sdp-func/math/sdp_math_macros.h"
#include "ska-sdp-func/utility/sdp_device_wrapper.h"
#include "ska-sdp-func/utility/sdp_mem_view.h"


template<typename UVW_TYPE>
__global__ void sdp_gridder_clamp_channels_single(
        sdp_MemViewGpu<const UVW_TYPE, 2> uvws,
        const int dim,
        const double freq0_hz,
        const double dfreq_hz,
        sdp_MemViewGpu<const int, 1> start_ch_in,
        sdp_MemViewGpu<const int, 1> end_ch_in,
        const double min_u,
        const double max_u,
        sdp_MemViewGpu<int, 1> start_ch_out,
        sdp_MemViewGpu<int, 1> end_ch_out
)
{
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t num_uvw = uvws.shape[0];
    if (i >= num_uvw) return;
    const double u0 = uvws(i, dim) * (freq0_hz / C_0);
    const double du = uvws(i, dim) * (dfreq_hz / C_0);
    const double rel_min_u = fabs(min_u - u0);
    const double rel_max_u = fabs(max_u - u0);
    const double eta_u = MAX(rel_min_u, rel_max_u) / 2147483645.0;
    if (fabs(du) > eta_u) // Use a safe value for eta.
    {
        const int64_t mins = (int64_t) (ceil((min_u - u0) / du));
        const int64_t maxs = (int64_t) (ceil((max_u - u0) / du));
        const int is_positive = du > 0;
        const int start_ch_ = is_positive ? (int) mins : (int) maxs;
        const int end_ch_ = is_positive ? (int) maxs : (int) mins;
        start_ch_out(i) = MAX(start_ch_in(i), start_ch_);
        end_ch_out(i) = MIN(end_ch_in(i), end_ch_);
    }
    else if (min_u > u0 || max_u <= u0)
    {
        start_ch_out(i) = 0;
        end_ch_out(i) = 0;
    }
    else
    {
        start_ch_out(i) = start_ch_in(i);
        end_ch_out(i) = end_ch_in(i);
    }
    end_ch_out(i) = MAX(end_ch_out(i), start_ch_out(i));
}


template<typename UVW_TYPE>
__global__ void sdp_gridder_clamp_channels_uv(
        sdp_MemViewGpu<const UVW_TYPE, 2> uvws,
        const double freq0_hz,
        const double dfreq_hz,
        sdp_MemViewGpu<const int, 1> start_ch_in,
        sdp_MemViewGpu<const int, 1> end_ch_in,
        const double min_u,
        const double max_u,
        const double min_v,
        const double max_v,
        sdp_MemViewGpu<int, 1> start_ch_out,
        sdp_MemViewGpu<int, 1> end_ch_out
)
{
    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t num_uvw = uvws.shape[0];
    if (i >= num_uvw) return;
    const double u0 = uvws(i, 0) * (freq0_hz / C_0);
    const double du = uvws(i, 0) * (dfreq_hz / C_0);
    const double rel_min_u = fabs(min_u - u0);
    const double rel_max_u = fabs(max_u - u0);
    const double eta_u = MAX(rel_min_u, rel_max_u) / 2147483645.0;
    if (fabs(du) > eta_u) // Use a safe value for eta.
    {
        const int64_t mins = (int64_t) (ceil((min_u - u0) / du));
        const int64_t maxs = (int64_t) (ceil((max_u - u0) / du));
        const int is_positive = du > 0;
        const int start_ch_ = is_positive ? (int) mins : (int) maxs;
        const int end_ch_ = is_positive ? (int) maxs : (int) mins;
        start_ch_out(i) = MAX(start_ch_in(i), start_ch_);
        end_ch_out(i) = MIN(end_ch_in(i), end_ch_);
    }
    else if (min_u > u0 || max_u <= u0)
    {
        start_ch_out(i) = 0;
        end_ch_out(i) = 0;
    }
    else
    {
        start_ch_out(i) = start_ch_in(i);
        end_ch_out(i) = end_ch_in(i);
    }
    end_ch_out(i) = MAX(end_ch_out(i), start_ch_out(i));
    if (start_ch_out(i) >= end_ch_out(i)) return;

    const double v0 = uvws(i, 1) * (freq0_hz / C_0);
    const double dv = uvws(i, 1) * (dfreq_hz / C_0);
    const double rel_min_v = fabs(min_v - v0);
    const double rel_max_v = fabs(max_v - v0);
    const double eta_v = MAX(rel_min_v, rel_max_v) / 2147483645.0;
    if (fabs(dv) > eta_v) // Use a safe value for eta.
    {
        const int64_t mins = (int64_t) (ceil((min_v - v0) / dv));
        const int64_t maxs = (int64_t) (ceil((max_v - v0) / dv));
        const int is_positive = dv > 0;
        const int start_ch_ = is_positive ? (int) mins : (int) maxs;
        const int end_ch_ = is_positive ? (int) maxs : (int) mins;
        start_ch_out(i) = MAX(start_ch_in(i), start_ch_);
        end_ch_out(i) = MIN(end_ch_in(i), end_ch_);
    }
    else if (min_v > v0 || max_v <= v0)
    {
        start_ch_out(i) = 0;
        end_ch_out(i) = 0;
    }
    else
    {
        start_ch_out(i) = start_ch_in(i);
        end_ch_out(i) = end_ch_in(i);
    }
    end_ch_out(i) = MAX(end_ch_out(i), start_ch_out(i));
}

SDP_CUDA_KERNEL(sdp_gridder_clamp_channels_single<float>)
SDP_CUDA_KERNEL(sdp_gridder_clamp_channels_single<double>)

SDP_CUDA_KERNEL(sdp_gridder_clamp_channels_uv<float>)
SDP_CUDA_KERNEL(sdp_gridder_clamp_channels_uv<double>)
