#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */

#include <thrust/complex.h>

#include "ska-sdp-func/grid_data/sdp_gridder_clamp_channels.h"
#include "ska-sdp-func/utility/sdp_cuda_atomics.h"
#include "ska-sdp-func/utility/sdp_device_wrapper.h"
#include "ska-sdp-func/utility/sdp_mem_view.h"

using thrust::complex;


template<typename UVW_TYPE, typename VIS_TYPE>
__global__ void sdp_gridder_wtower_degrid(
        const VIS_TYPE* const __restrict__ subgrids, // internal data
        const int w_plane,
        const int subgrid_offset_u,
        const int subgrid_offset_v,
        const int subgrid_offset_w,
        const double freq0_hz,
        const double dfreq_hz,
        const sdp_MemViewGpu<const UVW_TYPE, 2> uvws, // external data
        const sdp_MemViewGpu<const int, 1> start_chs, // external data
        const sdp_MemViewGpu<const int, 1> end_chs, // external data
        const double* const __restrict__ uv_kernel, // internal data
        const double* const __restrict__ w_kernel, // internal data
        const int subgrid_size,
        const int vr_size,
        const int support,
        const int w_support,
        const int oversample,
        const int w_oversample,
        const double theta,
        const double w_step,
        sdp_MemViewGpu<VIS_TYPE, 2> vis // external data
)
{
    const int64_t i_row = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t num_uvw = uvws.shape[0];
    if (i_row >= num_uvw)
        return;

    // Each row contains visibilities for all channels.
    // Skip if there's no visibility to degrid.
    int64_t start_ch = start_chs(i_row), end_ch = end_chs(i_row);
    if (start_ch >= end_ch)
        return;

    // Select only visibilities on this w-plane.
    const UVW_TYPE uvw[] = {uvws(i_row, 0), uvws(i_row, 1), uvws(i_row, 2)};
    const double min_w = (w_plane + subgrid_offset_w - 1) * w_step;
    const double max_w = (w_plane + subgrid_offset_w) * w_step;
    sdp_gridder_clamp_channels_inline(
            uvw[2], freq0_hz, dfreq_hz, &start_ch, &end_ch, min_w, max_w
    );
    if (start_ch >= end_ch)
        return;

    // Scale + shift UVWs.
    const double s_uvw0 = freq0_hz / C_0, s_duvw = dfreq_hz / C_0;
    double uvw0[] = {uvw[0] * s_uvw0, uvw[1] * s_uvw0, uvw[2] * s_uvw0};
    double duvw[] = {uvw[0] * s_duvw, uvw[1] * s_duvw, uvw[2] * s_duvw};
    uvw0[0] -= subgrid_offset_u / theta;
    uvw0[1] -= subgrid_offset_v / theta;
    uvw0[2] -= ((subgrid_offset_w + w_plane) * w_step);

    const int64_t subgrid_square = subgrid_size * subgrid_size;
    const int half_subgrid = subgrid_size / 2;
    const double half_vr_m1 = (vr_size - 1) / 2.0;
    const int half_vr = vr_size / 2;

    // Loop over selected channels.
    for (int64_t c = start_ch; c < end_ch; c++)
    {
        const double u = uvw0[0] + c * duvw[0];
        const double v = uvw0[1] + c * duvw[1];
        const double w = uvw0[2] + c * duvw[2];

        // Determine top-left corner of grid region
        // centered approximately on visibility.
        const int iu0 = int(round(theta * u - half_vr_m1)) + half_subgrid;
        const int iv0 = int(round(theta * v - half_vr_m1)) + half_subgrid;
        const int iu_shift = iu0 + half_vr - half_subgrid;
        const int iv_shift = iv0 + half_vr - half_subgrid;

        // Determine which kernel to use.
        int u_off = int(round((u * theta - iu_shift + 1) * oversample));
        int v_off = int(round((v * theta - iv_shift + 1) * oversample));
        int w_off = int(round((w / w_step + 1) * w_oversample));

        // Cater for the negative indexing which is allowed in Python!
        if (u_off < 0) u_off += oversample + 1;
        if (v_off < 0) v_off += oversample + 1;
        if (w_off < 0) w_off += w_oversample + 1;
        u_off *= support;
        v_off *= support;
        w_off *= w_support;

        // Degrid visibility.
        VIS_TYPE local_vis = (VIS_TYPE) 0;
        for (int iw = 0; iw < w_support; ++iw)
        {
            const double kern_w = w_kernel[w_off + iw];
            for (int iu = 0; iu < support; ++iu)
            {
                const double kern_wu = kern_w * uv_kernel[u_off + iu];
                for (int iv = 0; iv < support; ++iv)
                {
                    const double kern_wuv = kern_wu * uv_kernel[v_off + iv];
                    int ix_u = iu0 + iu;
                    int ix_v = iv0 + iv;
                    if (ix_u < 0) ix_u += subgrid_size;
                    if (ix_v < 0) ix_v += subgrid_size;
                    const int64_t idx = (
                        iw * subgrid_square + ix_u * subgrid_size + ix_v
                    );
                    local_vis += ((VIS_TYPE) kern_wuv * subgrids[idx]);
                }
            }
        }
        vis(i_row, c) += local_vis;
    }
}


template<typename UVW_TYPE, typename VIS_TYPE>
__global__ void sdp_gridder_wtower_grid(
        VIS_TYPE* __restrict__ subgrids, // internal data
        const int w_plane,
        const int subgrid_offset_u,
        const int subgrid_offset_v,
        const int subgrid_offset_w,
        const double freq0_hz,
        const double dfreq_hz,
        const sdp_MemViewGpu<const UVW_TYPE, 2> uvws, // external data
        const sdp_MemViewGpu<const int, 1> start_chs, // external data
        const sdp_MemViewGpu<const int, 1> end_chs, // external data
        const double* const __restrict__ uv_kernel, // internal data
        const double* const __restrict__ w_kernel, // internal data
        const int subgrid_size,
        const int vr_size,
        const int support,
        const int w_support,
        const int oversample,
        const int w_oversample,
        const double theta,
        const double w_step,
        const sdp_MemViewGpu<const complex<VIS_TYPE>, 2> vis // external data
)
{
    const int64_t i_row = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t num_uvw = uvws.shape[0];
    if (i_row >= num_uvw)
        return;

    // Each row contains visibilities for all channels.
    // Skip if there's no visibility to grid.
    int64_t start_ch = start_chs(i_row), end_ch = end_chs(i_row);
    if (start_ch >= end_ch)
        return;

    // Select only visibilities on this w-plane.
    const UVW_TYPE uvw[] = {uvws(i_row, 0), uvws(i_row, 1), uvws(i_row, 2)};
    const double min_w = (w_plane + subgrid_offset_w - 1) * w_step;
    const double max_w = (w_plane + subgrid_offset_w) * w_step;
    sdp_gridder_clamp_channels_inline(
            uvw[2], freq0_hz, dfreq_hz, &start_ch, &end_ch, min_w, max_w
    );
    if (start_ch >= end_ch)
        return;

    // Scale + shift UVWs.
    const double s_uvw0 = freq0_hz / C_0, s_duvw = dfreq_hz / C_0;
    double uvw0[] = {uvw[0] * s_uvw0, uvw[1] * s_uvw0, uvw[2] * s_uvw0};
    double duvw[] = {uvw[0] * s_duvw, uvw[1] * s_duvw, uvw[2] * s_duvw};
    uvw0[0] -= subgrid_offset_u / theta;
    uvw0[1] -= subgrid_offset_v / theta;
    uvw0[2] -= ((subgrid_offset_w + w_plane) * w_step);

    const int64_t subgrid_square = subgrid_size * subgrid_size;
    const int half_subgrid = subgrid_size / 2;
    const double half_vr_m1 = (vr_size - 1) / 2.0;
    const int half_vr = vr_size / 2;

    // Loop over selected channels.
    for (int64_t c = start_ch; c < end_ch; c++)
    {
        const double u = uvw0[0] + c * duvw[0];
        const double v = uvw0[1] + c * duvw[1];
        const double w = uvw0[2] + c * duvw[2];

        // Determine top-left corner of grid region
        // centered approximately on visibility.
        const int iu0 = int(round(theta * u - half_vr_m1)) + half_subgrid;
        const int iv0 = int(round(theta * v - half_vr_m1)) + half_subgrid;
        const int iu_shift = iu0 + half_vr - half_subgrid;
        const int iv_shift = iv0 + half_vr - half_subgrid;

        // Determine which kernel to use.
        int u_off = int(round((u * theta - iu_shift + 1) * oversample));
        int v_off = int(round((v * theta - iv_shift + 1) * oversample));
        int w_off = int(round((w / w_step + 1) * w_oversample));

        // Cater for the negative indexing which is allowed in Python!
        if (u_off < 0) u_off += oversample + 1;
        if (v_off < 0) v_off += oversample + 1;
        if (w_off < 0) w_off += w_oversample + 1;
        u_off *= support;
        v_off *= support;
        w_off *= w_support;

        // Grid visibility.
        const complex<double> local_vis = vis(i_row, c);
        for (int iw = 0; iw < w_support; ++iw)
        {
            const double kern_w = w_kernel[w_off + iw];
            for (int iu = 0; iu < support; ++iu)
            {
                const double kern_wu = kern_w * uv_kernel[u_off + iu];
                for (int iv = 0; iv < support; ++iv)
                {
                    const double kern_wuv = kern_wu * uv_kernel[v_off + iv];
                    int ix_u = iu0 + iu;
                    int ix_v = iv0 + iv;
                    if (ix_u < 0) ix_u += subgrid_size;
                    if (ix_v < 0) ix_v += subgrid_size;
                    const int64_t idx = 2 * (
                        iw * subgrid_square + ix_u * subgrid_size + ix_v
                    );
                    const complex<double> grid_val = (
                        (complex<double>) kern_wuv * local_vis
                    );
                    // The atomic adds will be very slow.
                    sdp_atomic_add(&subgrids[idx],     grid_val.real());
                    sdp_atomic_add(&subgrids[idx + 1], grid_val.imag());
                }
            }
        }
    }
}


SDP_CUDA_KERNEL(sdp_gridder_wtower_degrid<double, complex<double> >)
SDP_CUDA_KERNEL(sdp_gridder_wtower_degrid<float, complex<float> >)
SDP_CUDA_KERNEL(sdp_gridder_wtower_grid<double, double>)
SDP_CUDA_KERNEL(sdp_gridder_wtower_grid<float, float>)
