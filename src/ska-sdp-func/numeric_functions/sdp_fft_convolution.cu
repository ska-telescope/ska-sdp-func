#include "hip/hip_runtime.h"
/* See the LICENSE file at the top-level directory of this distribution. */

#include "ska-sdp-func/utility/sdp_device_wrapper.h"
#include "ska-sdp-func/fourier_transforms/sdp_fft.h"
#include"ska-sdp-func/utility/sdp_mem.h"

#include <hip/hip_complex.h>

template<typename T>
__global__ void pad_2D_gpu(
        const T *data,
        T *padded_data,
        int64_t rows,
        int64_t cols,
        int64_t pad_rows,
        int64_t pad_cols,
        int64_t padded_cols) {

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        padded_data[(i+pad_rows)*padded_cols + (j+pad_cols)] = data[i*cols + j];
    }
}

SDP_CUDA_KERNEL(pad_2D_gpu<hipDoubleComplex>);
SDP_CUDA_KERNEL(pad_2D_gpu<hipFloatComplex>);

template<typename T>
__global__ void complex_multiply(
        const T* in1,
        const T* in2,
        T* out,
        int64_t size) {

}

template<>
__global__ void complex_multiply<hipDoubleComplex>(
        const hipDoubleComplex* in1,
        const hipDoubleComplex* in2,
        hipDoubleComplex* out,
        int64_t size) {

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        out[i] = hipCmul(in1[i], in2[i]);
    }
}

template<>
__global__ void complex_multiply<hipFloatComplex>(
        const hipFloatComplex* in1,
        const hipFloatComplex* in2,
        hipFloatComplex* out,
        int64_t size) {

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        out[i] = hipCmulf(in1[i], in2[i]);
    }
}

SDP_CUDA_KERNEL(complex_multiply<hipDoubleComplex>);
SDP_CUDA_KERNEL(complex_multiply<hipFloatComplex>);

template<typename T>
__global__ void fft_normalise_gpu(
        T* fft_in,
        int64_t size){

}

template<>
__global__ void fft_normalise_gpu<hipDoubleComplex>(
        hipDoubleComplex* fft_in,
        int64_t size){

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    hipDoubleComplex normalise = make_hipDoubleComplex(size,0);

    if (i < size) {
        fft_in[i] = hipCdiv(fft_in[i], normalise);
    }
}

template<>
__global__ void fft_normalise_gpu<hipFloatComplex>(
        hipFloatComplex* fft_in,
        int64_t size){

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    hipFloatComplex normalise = make_hipFloatComplex(size,0);

    if (i < size) {
        fft_in[i] = hipCdivf(fft_in[i], normalise);
    }
}

SDP_CUDA_KERNEL(fft_normalise_gpu<hipDoubleComplex>);
SDP_CUDA_KERNEL(fft_normalise_gpu<hipFloatComplex>);

template<typename T>
__global__ void fft_shift_2D_gpu(
        T *data,
        T *shifted_data,
        int64_t rows,
        int64_t cols,
        int64_t half_rows,
        int64_t half_cols) {

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < half_rows && j < half_cols) {
            // bottom right to top left
            shifted_data[i*cols + j] = data[(i+half_rows)*cols + j + half_cols];
            // top right to bottom left
            shifted_data[(i+half_rows)*cols + j] = data[i*cols + j + half_cols];
            // top left to bottom right
            shifted_data[(i+half_rows)*cols + j + half_cols] = data[i*cols + j];
            // bottom left to top right
            shifted_data[i*cols + j + half_cols] = data[(i+half_rows)*cols + j];
    }
}
SDP_CUDA_KERNEL(fft_shift_2D_gpu<hipDoubleComplex>);
SDP_CUDA_KERNEL(fft_shift_2D_gpu<hipFloatComplex>);

template<typename T>
__global__ void remove_padding_2D_gpu(
        T *padded_data,
        T *data,
        int64_t rows,
        int64_t cols,
        int64_t pad_rows,
        int64_t pad_cols,
        int64_t original_rows,
        int64_t original_cols) {

    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < original_rows && j < original_cols) {
        data[i*original_cols + j] = padded_data[(i+(pad_rows-1))*cols + (j+(pad_cols-1))];
    }
}

SDP_CUDA_KERNEL(remove_padding_2D_gpu<hipDoubleComplex>);
SDP_CUDA_KERNEL(remove_padding_2D_gpu<hipFloatComplex>);